#include "hip/hip_runtime.h"

#include "common.h"
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "sensor.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) 
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 color(const ray& r, hitable** world)
{
    hit_record rec;
    if ((*world)->hit(r, 0.0, FLT_MAX, rec))
    {
        return 0.5f * vec3(rec.normal.x() + 1.0f, rec.normal.y() + 1.0f, rec.normal.z() + 1.0f);
    }
    else
    {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5f * (unit_direction.y() + 1.0f);
        return (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

/* TODO: Move the random seed initialization to this function as:
hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);*/
__global__ void render(vec3* fb, int max_x, int max_y, int ns,
    camera **cam, hitable **world, hiprandState *rand_state) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);
    for (int s = 0; s < ns; s++) 
    {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v);
        col += color(r, world);
    }
    fb[pixel_index] = col / float(ns);
}

__global__ void create_world(hitable** d_list, hitable** d_world, camera** d_camera)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list) = new sphere(vec3(0, 0, -1), 0.5);
        *(d_list + 1) = new sphere(vec3(0, -100.5, -1), 100);
        *d_world = new hitable_list(d_list, 2);
        *d_camera = new camera();
    }
}

__global__ void free_world(hitable** d_list, hitable** d_world, camera** d_camera)
{
    delete* (d_list);
    delete* (d_list + 1);
    delete* d_world;
    delete* d_camera;
}

int main(int argc, char* argv[])
{
    int nx, ny, ns, tx, ty;
    /* TODO: Fiddle with tx/ty thread sizes to see what works best*/

    if (argc > 1)
    {
        nx = std::stoi(argv[1]);
        ny = std::stoi(argv[2]);
        ns = std::stoi(argv[3]);
        tx = std::stoi(argv[4]);
        ty = std::stoi(argv[5]);
    }
    else
    {
        std::cout << argv[0] << std::endl;
        nx = 1200;
        ny = 600;
        ns = 100;
        tx = 8;
        ty = 8;
    }

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate FB
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // allocate random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));

    // make our world of hitables & the camera
    hitable** d_list;
    checkCudaErrors(hipMalloc((void**)&d_list, 2 * sizeof(hitable*)));
    hitable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable*)));
    camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
    create_world << <1, 1 >> > (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state); // only exists to initialize rand numbers separately from render
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::ofstream img;
    img.open("C:\\Users\\areigle\\Documents\\raytracer\\RayTracing\\CudaWeekend\\images\\image.ppm", std::ios::in | std::ios::out);
    /* For later: The "Current directory" is the C drive - I'll need a better way to resolve the image file location */
    /* There's also the bug that a file named "image.ppm" must exist prior to opening - esay fix but I'm lazy*/

    if (!img.is_open())
    {
        std::cout << "Error: Invalid File" << std::endl;
        return 1;
    }

    img << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            img << ir << " " << ig << " " << ib << "\n";
        }
    }

    img.close();

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world << <1, 1 >> > (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));
    

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
}
